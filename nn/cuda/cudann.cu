#include "hip/hip_runtime.h"
#include "cudann.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

__device__ float sigmoid(float x) {
	return 1.0 / (1.0 + expf(-x));
}

__device__ float sigmoid_gradient(float x) {
	return x * (1 - x);
}

__global__ void
forward_d(f16* up, int num_up, f16* down, int num_down, f16* param) {
	int upI = blockDim.x * blockIdx.x + threadIdx.x;
	float v = 0;
	for (int downI = 0; downI < num_down; downI++) {
		float p = __half2float(param[upI*num_up+downI]);
		float d = __half2float(down[downI]);
		v += p * d;
	}
	v = (v>0) ? v : 0;
	//v = sigmoid(v);
	up[upI] = __float2half_rn(v);
}

void forward(f16* up, int num_up, f16* down, int num_down, f16* param) {
	int threads = 8;
	int blocks = (num_up + threads - 1) / threads;
	forward_d<<<blocks, threads>>>(up, num_up, down, num_down, param);
}

__global__ void
backward_d(f16* up, f16* up_err, int num_up, f16* down, f16* down_err, int num_down, f16* param) {
	int downI = blockDim.x * blockIdx.x + threadIdx.x;

	float v = 0;
	for (int upI = 0; upI < num_up; upI++) {
		int paramI = upI*num_up+downI;
		float p = __half2float(param[paramI]);
		float u = __half2float(up_err[upI]);
		v += p * u;
	}
	down_err[downI] = __float2half_rn(v);

	float down_orig = __half2float(down[downI]);
	for (int upI = 0; upI < num_up; upI++) {
		float up_orig = __half2float(up[upI]);
		//float gradient = sigmoid_gradient(up_orig);
		float gradient = 1;
		float delta = __half2float(up_err[upI]) * gradient * down_orig;
		int paramI = upI*num_up+downI;
		float p = __half2float(param[paramI]);
		p = (p>0) ? p : 0; // max(0, v)
		param[paramI] = __float2half_rn(p + delta);
	}
}

void backward(f16* up, f16* up_err, int num_up, f16* down, f16* down_err, int num_down, f16* param) {
	int threads = 256;
	int blocks = (num_down + threads - 1) / threads;
	backward_d<<<blocks, threads>>>(up, up_err, num_up, down, down_err, num_down, param);
}

__global__ void f16devsub_d(f16* dst, f16* a, f16* b, int count) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	float af = __half2float(a[i]);
	float bf = __half2float(b[i]);
	dst[i] = __float2half_rn(af - bf);
}

void f16devsub(f16* dst, f16* a, f16* b, int count) {
	int threads = 128;
	int blocks = (count + threads - 1) / threads;
	f16devsub_d<<<blocks, threads>>>(dst, a, b, count);
}

f16* alloc_f16_device(int count) {
	f16* f;
	hipError_t err = hipMalloc((void**)&f, count*sizeof(f16));
	if (err != hipSuccess) {
		fprintf(stderr, "dalloc_f16(%d): %s", count, hipGetErrorString(err));
		exit(1);
	}
	return f;
}

void memcpy_htod(f16* d, const f16* h, int count) {
	hipError_t err = hipMemcpy(d, h, count*sizeof(f16), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "memcpy_htod: %s", hipGetErrorString(err));
		exit(1);
	}
	printf("hi\n");
}

void memcpy_dtoh(f16* h, const f16* d, int count) {
	hipError_t err = hipMemcpy(h, d, count*sizeof(f16), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "memcpy_dtoh: %s", hipGetErrorString(err));
		exit(1);
	}
}
