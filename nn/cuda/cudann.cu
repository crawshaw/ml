#include "hip/hip_runtime.h"
#include "cudann.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

__global__ void
forward_d(f16* up, int num_up, f16* down, int num_down, f16* param) {
	int upI = blockDim.x * blockIdx.x + threadIdx.x;
	float v = 0;
	for (int downI = 0; downI < num_down; downI++) {
		float p = __half2float(param[upI*num_up+downI]);
		float d = __half2float(down[downI]);
		v += p * d;
	}
	v = (v>0) ? v : 0;
	up[upI] = __float2half_rn(v);
}

void forward(f16* up, int num_up, f16* down, int num_down, f16* param) {
	int threads = 256;
	int blocks = (num_up + threads - 1) / threads;
	forward_d<<<blocks, threads>>>(up, num_up, down, num_down, param);
}

void backward(f16* up, f16* up_err, int num_up, f16* down, f16* down_err, int num_down, f16* param) {
}

f16* alloc_f16_device(int count) {
	f16* f;
	hipError_t err = hipMalloc((void**)&f, count*sizeof(f16));
	if (err != hipSuccess) {
		fprintf(stderr, "dalloc_f16(%d): %s", count, hipGetErrorString(err));
		exit(1);
	}
	return f;
}

void memcpy_htod(f16* d, const f16* h, int count) {
	hipError_t err = hipMemcpy(d, h, count*sizeof(f16), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "memcpy_htod: %s", hipGetErrorString(err));
		exit(1);
	}
	printf("hi\n");
}

void memcpy_dtoh(f16* h, const f16* d, int count) {
	hipError_t err = hipMemcpy(h, d, count*sizeof(f16), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "memcpy_dtoh: %s", hipGetErrorString(err));
		exit(1);
	}
}
